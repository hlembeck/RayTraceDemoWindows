#include "RayTraceEntry.cuh"

bool populateRays(ImgParamPinhole& params, Ray* &rays, hipError_t& cudaStatus);
bool populateIntersectionData(ImgParamPinhole& params, Ray* &rays, Face* faces, unsigned int numFaces, IntersectionData* &intersectionData, hipError_t& cudaStatus);
bool populateSamples(ImgParamPinhole& params, IntersectionData* &intersectionData, double* &samples, hipError_t& cudaStatus);
bool populatePixelSamples(ImgParamPinhole& params, double* &samples, double* &pixelSamples, hipError_t& cudaStatus);
bool populateRGBQuadArray(ImgParamPinhole& params, unsigned int rowWidth, unsigned char* &rgbQuadArr, double* pixelSamples, hipError_t& cudaStatus);

bool paramsToDevice(SceneParams& sceneParams, double*& spectrums, double*& spectrumsBack, Face*& faces, unsigned int*& meshes, hipError_t& cudaStatus);

bool rtWrapper(RTParams& rtParams) {
	RTParams rtBlockParams = rtParams;

	if (rtBlockParams.params.sensorWidth / (double)rtBlockParams.params.width != rtBlockParams.params.sensorHeight / (double)rtBlockParams.params.height)
		return false;

	std::pair<double, double> unitsPerPixel = { rtBlockParams.params.sensorWidth / rtBlockParams.params.width, rtBlockParams.params.sensorHeight / rtBlockParams.params.height };
	unsigned int stepsWidth = rtBlockParams.params.width / 256;
	unsigned int stepsHeight = rtBlockParams.params.height / 256;

	rtParams.rgbQuadArr = new unsigned char[4 * rtParams.params.width * rtParams.params.height];
	rtBlockParams.rgbQuadArr = rtParams.rgbQuadArr;

	if (stepsWidth) {
		rtBlockParams.params.sensorWidth = 256 * unitsPerPixel.first;
		rtBlockParams.params.width = 256;
		if (stepsHeight) {
			rtBlockParams.params.sensorHeight = 256 * unitsPerPixel.second;
			rtBlockParams.params.height = 256;
			//Fill blocks of size 256x256
			for (unsigned int i = 0; i < stepsHeight; i++) {
				rtBlockParams.params.left = rtParams.params.left;
				for (unsigned int j = 0; j < stepsWidth; j++) {
					//printf("(%g %g)[%g %g]\n\n\n", rtBlockParams.params.left, rtBlockParams.params.top, rtBlockParams.params.sensorWidth, rtBlockParams.params.sensorHeight);
					if (!getPinholeImage(rtBlockParams, rtParams.params.width * 4)) {
						OutputDebugString(TEXT("(In rtWrapper()) Failed getPinholeImage().\n"));
						return false;
					}
					rtBlockParams.rgbQuadArr += 1024;
					rtBlockParams.params.left += rtBlockParams.params.sensorWidth;
				}
				rtBlockParams.rgbQuadArr += 261120 * stepsWidth;
				rtBlockParams.params.top -= rtBlockParams.params.sensorHeight;
			}
			//Fill blocks on right side except bottom
			rtBlockParams.params.top = rtParams.params.top;
			rtBlockParams.params.width = rtParams.params.width - stepsWidth * 256;
			rtBlockParams.rgbQuadArr = rtParams.rgbQuadArr + 1024 * stepsWidth;
			rtBlockParams.params.sensorWidth = rtParams.params.sensorWidth - (stepsWidth * 256) * unitsPerPixel.first;
			for (unsigned int i = 0; i < stepsHeight; i++) {

				if (!getPinholeImage(rtBlockParams, rtParams.params.width * 4)) {
					OutputDebugString(TEXT("(In rtWrapper()) Failed getPinholeImage().\n"));
					return false;
				}

				rtBlockParams.rgbQuadArr += 1024 * rtParams.params.width;
				rtBlockParams.params.top -= rtBlockParams.params.sensorHeight;
			}
			//Fill blocks on the bottom, except the right corner
			rtBlockParams.params.left = rtParams.params.left;
			rtBlockParams.params.width = 256;
			rtBlockParams.params.sensorWidth = 256 * unitsPerPixel.first;
			rtBlockParams.params.height = rtParams.params.height - 256 * stepsHeight;
			rtBlockParams.params.sensorHeight = rtParams.params.sensorHeight - (256 * stepsHeight) * unitsPerPixel.second;
			rtBlockParams.rgbQuadArr = rtParams.rgbQuadArr + 1024 * stepsHeight * rtParams.params.width;
			for (unsigned int i = 0; i < stepsWidth; i++) {
				if (!getPinholeImage(rtBlockParams, rtParams.params.width * 4)) {
					OutputDebugString(TEXT("(In rtWrapper()) Failed getPinholeImage().\n"));
					return false;
				}

				rtBlockParams.rgbQuadArr += 1024;
				rtBlockParams.params.left += rtBlockParams.params.sensorWidth;
			}
			//Fill bottom-right corner
			rtBlockParams.params.width = rtParams.params.width - 256 * stepsWidth;
			rtBlockParams.params.sensorWidth = rtParams.params.sensorWidth - (stepsWidth * 256) * unitsPerPixel.first;
			if (!getPinholeImage(rtBlockParams, rtParams.params.width * 4)) {
				OutputDebugString(TEXT("(In rtWrapper()) Failed getPinholeImage().\n"));
				return false;
			}
			return true;
		}
		//Fill row
		rtBlockParams.params.left = rtParams.params.left;
		rtBlockParams.params.width = 256;
		rtBlockParams.params.sensorWidth = 256 * unitsPerPixel.first;
		for (unsigned int i = 0; i < stepsWidth; i++) {
			if (!getPinholeImage(rtBlockParams, rtParams.params.width * 4)) {
				OutputDebugString(TEXT("(In rtWrapper()) Failed getPinholeImage().\n"));
				return false;
			}
			rtBlockParams.rgbQuadArr += 1024;
			rtBlockParams.params.left += rtBlockParams.params.sensorWidth;
		}
		return true;
	}
	else if (stepsHeight) {
		//Fill column
		rtBlockParams.params.sensorHeight = 256 * unitsPerPixel.second;
		rtBlockParams.params.height = 256;
		rtBlockParams.params.width = rtParams.params.width - stepsWidth * 256;
		rtBlockParams.params.sensorWidth = rtParams.params.sensorWidth - (stepsWidth * 256) * unitsPerPixel.first;
		for (unsigned int i = 0; i < stepsHeight; i++) {
			if (!getPinholeImage(rtBlockParams, rtParams.params.width * 4)) {
				OutputDebugString(TEXT("(In rtWrapper()) Failed getPinholeImage().\n"));
				return false;
			}
			rtBlockParams.rgbQuadArr += 1024 * rtParams.params.width;
			rtBlockParams.params.top -= rtBlockParams.params.sensorHeight;
		}
		return true;
	}

	if (!getPinholeImage(rtParams, rtParams.params.width * 4)) {
		OutputDebugString(TEXT("(In rtWrapper()) Failed getPinholeImage().\n"));
		return false;
	}

	return true;
}

bool getPinholeImage(RTParams& rtBlockParams, unsigned int rowWidth) {
	/*
	To store the ray information for each sample ray.
	LOCATION: Device
	LENGTH: width * height * nRays * nRays
	*/
	Ray* rays = 0;
	/*
	To store the ray information for each refracted ray.
	LOCATION: Device
	LENGTH: width * height * nRays * nRays * 
	*/
	Ray* refractedRays = 0;
	/*
	To store the spectrumand angle corresponding to the surface hit on the all reflections of all sample rays.
	LOCATION: Device
	LENGTH: params.width * params.height * nRays * nRays * nReflections
	*/
	IntersectionData* intersectionData = 0;
	/*
	To store the raw spectral data computed from intersectionData.
	LOCATION: Device
	LENGTH: width * height * nRays * nRays
	*/
	double* samples = 0;
	/*
	To store the spectra data for each pixel, as computed from samples.
	LOCATION: Device
	LENGTH: width * height
	*/
	double* pixelSamples = 0;
	/*
	To store the spectrums that the meshes reference.
	LOCATION: Values stored on Device
	*/
	double* spectrums = 0;
	double* spectrumsBack = 0;
	/*
	Array of Face structs that represents the faces present in a scene.
	LOCATION: Device.
	*/
	Face* faces = 0;
	/*
	Array of indices of faces that partitions faces into meshes.
	LOCATION: Device
	*/
	unsigned int* meshes = 0;
	//unsigned int numSpectrums = rtBlockParams.sceneParams.spectrums.size();

	hipError_t cudaStatus;

	hipSetDevice(0);

	if (!paramsToDevice(rtBlockParams.sceneParams, spectrums, spectrumsBack, faces, meshes, cudaStatus))
		goto error;

	//Populate rays with device-allocated data
	if (!populateRays(rtBlockParams.params, rays, cudaStatus))
		goto error;

	//Populate the intersection data
	if (!populateIntersectionData(rtBlockParams.params, rays, faces, rtBlockParams.sceneParams.faces.size(), intersectionData, cudaStatus))
		goto error;

	cudaStatus = hipFree(rays);
	cudaStatus = hipFree(faces);
	cudaStatus = hipFree(meshes);
	if (cudaStatus != hipSuccess) {
		printf("Error freeing rays, faces, or meshes.\n");
		goto error;
	}

	//Populate samples with data from intersectionData
	if (!populateSamples(rtBlockParams.params, intersectionData, samples, cudaStatus))
		goto error;

	cudaStatus = hipFree(spectrums);
	cudaStatus = hipFree(spectrumsBack);
	cudaStatus = hipFree(intersectionData);
	if (cudaStatus != hipSuccess) {
		printf("Failed to free intersectionData or spectrums.\n");
		goto error;
	}

	//Populate pixel samples using data from samples
	if (!populatePixelSamples(rtBlockParams.params, samples, pixelSamples, cudaStatus))
		goto error;

	cudaStatus = hipFree(samples);
	if (cudaStatus != hipSuccess) {
		printf("Failed to free samples.\n");
		goto error;
	}

	//populate rgbBMP with RGB data from pixelSamples. pixelSamples is freed in this function. rgbBMP is allocated in this function, and must be freed from the heap after painting in window.
	if (!populateRGBQuadArray(rtBlockParams.params, rowWidth, rtBlockParams.rgbQuadArr, pixelSamples, cudaStatus))
		goto error;

	hipDeviceReset();

	return true;
error:
	OutputDebugString(TEXT("(In getPinholeImage()) There was an error.\n"));
	hipFree(rays);
	hipFree(intersectionData);
	hipFree(spectrums);
	hipFree(spectrumsBack);
	hipFree(samples);
	hipFree(pixelSamples);
	hipDeviceReset();
	return false;
}

bool populateRays(ImgParamPinhole& params, Ray* &rays, hipError_t& cudaStatus) {
	cudaStatus = hipMalloc((void**)&rays, sizeof(Ray) * params.width * params.height * params.nRays * params.nRays);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRays()) Failed to allocate rays.\n"));
		return false;
	}

	dim3 numBlocks(params.height, params.width);
	dim3 numThreadsPerBlock(params.nRays, params.nRays);

	generateRaysPinhole << <numBlocks, numThreadsPerBlock >> > (rays, params.pinhole, params.top, params.left, params.sensorWidth / (double)params.width, params.sensorWidth / ((double)params.width * (double)params.nRays), params.width, params.height, params.nRays);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRays()) Failed generateRaysPinhole().\n"));
		return false;
	}
	return true;
}

bool populateIntersectionData(ImgParamPinhole& params, Ray* &rays, Face* faces, unsigned int numFaces, IntersectionData* &intersectionData, hipError_t& cudaStatus) {
	unsigned int planeLength = params.width * params.height * params.nRays * params.nRays;
	cudaStatus = hipMalloc((void**)&intersectionData, sizeof(IntersectionData) * params.width * params.height * params.nRays * params.nRays * (params.nReflections + 1));
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateIntersectionData()) Failed malloc of intersectionData.\n"));
		return false;
	}

	dim3 numBlocks(params.height, params.width);
	dim3 numThreadsPerBlock(params.nRays, params.nRays);
	for (unsigned int i = 0; i < params.nReflections + 1; i++) {
		traceRays << <numBlocks, numThreadsPerBlock >> > (rays, faces, numFaces, intersectionData + i, params.width, params.nRays, params.nReflections + 1);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			TCHAR buf[256] = L"";
			memset(buf, 0, sizeof(TCHAR) * 256);
			wsprintf(buf, L"(In populateIntersectionData()) Failed traceRays() on iteration %d.\n", i);
			OutputDebugString(buf);
			return false;
		}
		//printf("\n\n\n");
		/*printRays << <1, 1 >> > (rays, params.height * params.width * params.nRays * params.nRays);
		printf("\n\n\n\n");*/
	}
	return true;
}

bool populateSamples(ImgParamPinhole& params, IntersectionData* &intersectionData, double* &samples, hipError_t& cudaStatus) {
	cudaStatus = hipMalloc((void**)&samples, sizeof(double) * params.width * params.height * params.nRays * params.nRays * STEPS);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateSamples()) Failed malloc of samples.\n"));
		return false;
	}

	dim3 numBlocks(params.height, params.width);
	dim3 numThreadsPerBlock(params.nRays, params.nRays);

	computeSamples << <numBlocks, numThreadsPerBlock >> > (intersectionData, samples, params.width, params.nRays, params.nReflections + 1);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateSamples()) Failed computeSamples.\n"));
		return false;
	}
	return true;
}

bool populatePixelSamples(ImgParamPinhole& params, double* &samples, double* &pixelSamples, hipError_t& cudaStatus) {
	cudaStatus = hipMalloc((void**)&pixelSamples, sizeof(double) * params.width * params.height * STEPS);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populatePixelSamples()) Failed to allocate pixelSamples.\n"));
		return false;
	}
	dim3 numBlocks(params.height, params.width);
	computePixels << <numBlocks, 1 >> > (samples, pixelSamples, params.width, params.nRays);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populatePixelSamples()) Failed computePixels().\n"));
		return false;
	}
}

bool populateRGBQuadArray(ImgParamPinhole& params, unsigned int rowWidth, unsigned char* &rgbQuadArr, double* pixelSamples, hipError_t& cudaStatus) {

	double* colorMatchXYZ = 0;
	double* tristimulusPixels = 0;
	unsigned char* rgbPixels = 0;

	cudaStatus = hipMalloc((void**)&colorMatchXYZ, sizeof(double) * 3 * STEPS);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed malloc of colorMatchXYZ.\n"));
		return false;
	}

	getColorMatchXYZ << <3, STEPS >> > (colorMatchXYZ);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed getColorMatchXYZ().\n"));
		hipFree(colorMatchXYZ);
		hipFree(pixelSamples);
		return false;
	}

	cudaStatus = hipMalloc((void**)&tristimulusPixels, sizeof(double) * 3 * params.width * params.height);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed malloc of tristimulusPixels.\n"));
		hipFree(colorMatchXYZ);
		hipFree(pixelSamples);
		return false;
	}

	dim3 numBlocks(params.height, params.width);

	getTristimulus << <numBlocks, 3 >> > (pixelSamples, colorMatchXYZ, tristimulusPixels, params.width);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed getTristimulus().\n"));
		hipFree(colorMatchXYZ);
		hipFree(tristimulusPixels);
		hipFree(pixelSamples);
		return false;
	}

	cudaStatus = hipFree(pixelSamples);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed free of pixelSamples.\n"));
		return false;
	}

	cudaStatus = hipFree(colorMatchXYZ);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed free of colorMatchXYZ.\n"));
		return false;
	}

	cudaStatus = hipMalloc((void**)&rgbPixels, 4 * params.width * params.height);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed malloc of rgbPixels.\n"));
		hipFree(colorMatchXYZ);
		hipFree(tristimulusPixels);
		hipFree(pixelSamples);
		return false;
	}

	getRGB << <numBlocks, 3 >> > (tristimulusPixels, rgbPixels, params.width);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed getRGB.\n"));
		hipFree(colorMatchXYZ);
		hipFree(tristimulusPixels);
		hipFree(pixelSamples);
		hipFree(rgbPixels);
		return false;
	}

	cudaStatus = hipFree(tristimulusPixels);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed free of tristimulusPixels.\n"));
		return false;
	}

	for (unsigned int i = 0; i < params.height; i++) {
		cudaStatus = hipMemcpy(rgbQuadArr + i * rowWidth, rgbPixels + i * params.width * 4, 4 * params.width, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed memcopy of rgbPixels -> rgbBMP.\n"));
			hipFree(colorMatchXYZ);
			hipFree(tristimulusPixels);
			hipFree(pixelSamples);
			hipFree(rgbPixels);
			return false;
		}
	}

	cudaStatus = hipFree(rgbPixels);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In populateRGBQuadArray()) Failed free of rgbPixels.\n"));
		return false;
	}

	return true;
}

bool paramsToDevice(SceneParams& sceneParams, double*& spectrums, double*& spectrumsBack, Face*& faces, unsigned int*& meshes, hipError_t& cudaStatus) {
	unsigned int numBlocks;

	cudaStatus = hipMalloc((void**)&spectrums, sizeof(double) * sceneParams.spectrums.size());
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMalloc of spectrums.\n"));
		return false;
	}
	cudaStatus = hipMemcpy(spectrums, sceneParams.spectrums.data(), sizeof(double) * sceneParams.spectrums.size(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMemcpy of spectrums.\n"));
		return false;
	}

	cudaStatus = hipMalloc((void**)&spectrumsBack, sizeof(double) * sceneParams.spectrums.size());
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMalloc of spectrumsBack.\n"));
		return false;
	}
	cudaStatus = hipMemcpy(spectrumsBack, sceneParams.spectrumsBack.data(), sizeof(double) * sceneParams.spectrums.size(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMemcpy of spectrumsBack.\n"));
		return false;
	}

	cudaStatus = hipMalloc((void**)&faces, sizeof(Face) * sceneParams.faces.size());
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMalloc of faces.\n"));
		return false;
	}
	cudaStatus = hipMemcpy(faces, sceneParams.faces.data(), sizeof(Face) * sceneParams.faces.size(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMemcpy of faces.\n"));
		return false;
	}

	numBlocks = (sceneParams.faces.size() & 511 ? (sceneParams.faces.size() >> 9) + 1 : (sceneParams.faces.size() >> 9));
	setSpectrums << <numBlocks, 512 >> > (faces, sceneParams.faces.size(), spectrums, spectrumsBack);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed setSpectrums().\n"));
		return false;
	}

	cudaStatus = hipMalloc((void**)&meshes, sizeof(unsigned int) * sceneParams.meshes.size());
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMalloc of meshes.\n"));
		return false;
	}
	cudaStatus = hipMemcpy(meshes, &sceneParams.meshes[0], sizeof(unsigned int) * sceneParams.meshes.size(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		OutputDebugString(TEXT("(In paramsToDevice()) Failed hipMemcpy of meshes.\n"));
		return false;
	}

	return true;
}